#include <iostream>
#include "hip/hip_runtime.h"

#include <fstream>
#include <iomanip>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

using namespace std;
/*Kernel Function. Please Note that if you follow the code given below,
the number of threads and pixel are equal, i.e. each thread corresponds to pixels*/
__global__ void imageprocessing(int *pixeldata,int *pixelupdate, int height, int width)
{
	int offset = (blockIdx.x*gridDim.x+blockIdx.y*gridDim.y+threadIdx.x);
	pixelupdate[offset]=(int) (0.21f*pixeldata[offset]+0.71f*pixeldata[offset+width*height]+0.07f*pixeldata[offset+2*width*height]);
}

int main(){
	int height=300, width=400; //Enter the image height and width. For a 640x480 image, int height=480, width=640; 
	int thperblk=width/4;//Enter the number of threads per block
	int i,j,k;
	int block_dim_x, block_dim_y, grid_dim_x, grid_dim_y;

	int *pixeldata, *pixelupdate;
	FILE *data;
	pixeldata = (int*) malloc(height*width*3*sizeof(int));
	
	/*The size of memory allocated will be height*width*sizeof(int) 
	for grayscale output data as there is only one value per pixel*/
	pixelupdate = (int*) malloc(height*width*3*sizeof(int));
	
	data = fopen("pixeldata.txt","r");
	printf("starting\n");
	for(k=0; k<3; k++){
		for(j=0; j<width; j++){
			for(i=0; i<height; i++){
				fscanf(data, "%d", &pixeldata[k*width*height+j*height+i] );
			}
		}
	}
	fclose(data);
	printf("done\n");
	
	int *pixeldata_device;
	int *pixelupdate_device;
	int size = height*width*3*sizeof(int);
	
	hipMalloc((void**)&pixeldata_device, size);
	
	/*The size of memory allocated will be size/3 for grayscale 
	output data as there is only one value per pixel.
	Take care of this case in cudaMemcpy as well.*/
	hipMalloc((void**)&pixelupdate_device, size); 
	
	block_dim_x=thperblk; //Enter the number of threads in the x-direction in each block
	block_dim_y=thperblk/block_dim_x;
	grid_dim_x=height/block_dim_x;
	grid_dim_y=width/block_dim_y;
	dim3 threads(block_dim_x, block_dim_y, 1);
	dim3 blocks(grid_dim_x, grid_dim_y, 1);
	
	//Take care of the changes in size of array for grayscale case
	hipMemcpy(&pixeldata_device[0],&pixeldata[0],size,hipMemcpyHostToDevice);
	hipMemcpy(&pixelupdate_device[0],&pixeldata_device[0],size,hipMemcpyDeviceToDevice);
	imageprocessing <<<blocks, threads>>> (pixeldata_device,pixelupdate_device,height,width);
	hipMemcpy(&pixelupdate[0],&pixelupdate_device[0],size,hipMemcpyDeviceToHost);
	
	FILE *output;
	printf("open\n");
	output = fopen("pixeldataout.txt","w");
	
	//Outermost 'k' loop will not be present for gray scale case
	for(k=0; k<3; k++){
		for(i=0; i<width; i++){
			for(j=0; j<height; j++){
				fprintf(output, "%d\t", pixelupdate[i*height+j] );
			}
		}	
	}
	fclose(output);
	printf("close\n");
	hipFree(pixeldata_device);
	hipFree(pixelupdate_device);
	free(pixeldata);
	free(pixelupdate);
	return 0;
	}