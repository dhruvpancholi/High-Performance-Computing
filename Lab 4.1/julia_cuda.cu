/*Name: Dhruv Pancholi
* Roll No.: 11110028
* Julia Set on CUDA
*/
#include <X11/Xlib.h>
#include <X11/Xutil.h>
#include <X11/Xos.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include "stdlib.h"
#include "hip/hip_runtime.h"

#include<iostream>

#define DEBUG 0

typedef struct complextype
        {
        float real, imag;
        } Compl;

void compute(int X_RESN, int Y_RESN, int max_iteration);

/*This is the main routine which computes the color for each pixel in the screen.
The color is given in the variable <TheK>, which is obtained by iterating the
complex sequence Z = Z^2 + C 
*/
__global__ void iterate(int maxc,int minc, int X_RESN, int Y_RESN, int* TheK, int max_iteration)
{
	int i,j;
	int k, imax = max_iteration;
	float lengthsq, temp;
	Compl z,c;

	i = blockIdx.x*blockDim.x+threadIdx.x;
	j = blockIdx.y*blockDim.y+threadIdx.y;

	if (i<X_RESN && j<Y_RESN)
	{
		c.real = -0.8f; c.imag = 0.156f;
		z.real   = -2.0f + (4.0f/ (X_RESN-1) )*(float)j ;
		z.imag   = -2.0f - (4.0f/ (Y_RESN-1) )*( (float)i - (Y_RESN - 1) );
	}
	else{return;}
	

	k = 0;

	do  {       //iterate for pixel color 

	temp = z.real*z.real - z.imag*z.imag + c.real;
	z.imag = 2.0f*z.real*z.imag + c.imag;
	z.real = temp;
	lengthsq = z.real*z.real+z.imag*z.imag;
	k++;

	} while (lengthsq < 4.0f && k < imax);

	TheK[i*Y_RESN+j]=minc + ((float)(maxc - minc)/(imax -1 ))*( k -1 );

} // End of iterate


int main(int argc, char const *argv[]){
int max_iteration=200;
if (argv[1]!=NULL){
	max_iteration=atoi(argv[1]);
}
  if ( DEBUG  )
    printf("1. Entering code\n");

        int rank;
        int nproc;
        int             X_RESN = 800;
        int             Y_RESN = 800;
  
  if ( DEBUG  )
    printf("2. Initialized variables\n");
    
	clock_t t1, t2;
        t1=clock();
           compute(X_RESN,Y_RESN, max_iteration);
	t2=clock();   
  
  if ( DEBUG  ){printf("15. Finished Computation\n");}
    
	printf("Total Time Taken = %f Seconds\n",((double)( t2 - t1 )/ (double) CLOCKS_PER_SEC ) ); 

return 0;
}

void compute(int X_RESN, int Y_RESN, int max_iteration)
{
        Window          win;                            /* initialization for a window */
        unsigned
        int             width, height,                  /* window size */
                        x, y,                           /* window position */
                        border_width,                   /*border width in pixels */
                        display_width, display_height,  /* size of screen */
                        screen;                         /* which screen */

        char            *window_name = "Julia Set", *display_name = NULL;
        GC              gc;
        unsigned long   valuemask = 0;
        XGCValues       values;
        Display         *display;
        XSizeHints      size_hints;
        Pixmap          bitmap;
        XPoint          points[800];
        FILE            *fp, *fopen ();
        char            str[100];
        Status          rc;
        int             tmp=1;
        XSetWindowAttributes attr[1];
       
       if ( DEBUG  )
         printf("3. Finished Declaring X-Window parameters\n ");

       /* Important variables */
        int i, j, k;
        size_t size = (X_RESN*Y_RESN)*sizeof(int);
        int *TheK = (int *)malloc(size);
        int *TheK_device;
        hipMalloc((void **) &TheK_device, size);
       
        /* connect to Xserver */
        if ( DEBUG  )
         printf("4. Connecting to X-Server\n ");
	 
        if (  (display = XOpenDisplay (display_name)) == NULL ) {
           fprintf (stderr, "drawon: cannot connect to X server %s\n",
                                XDisplayName (display_name) );
          exit (-1);
        }
        
        /* get screen size */
        /*screen variable stands for which screen to be used*/
        screen = DefaultScreen (display);
        display_width = DisplayWidth (display, screen);
        display_height = DisplayHeight (display, screen);

        /* set window size */

        width = X_RESN;
        height = Y_RESN;

        /* set window position */

        x = 0;
        y = 0;

        /* create opaque window */
        if ( DEBUG  )
         printf("5. Creating an X-Window\n");
	  
        border_width = 4;
        win = XCreateSimpleWindow (display, RootWindow (display, screen),
                                x, y, width, height, border_width,
                                BlackPixel (display, screen), WhitePixel (display, screen));

        size_hints.flags = USPosition|USSize;
        size_hints.x = x;
        size_hints.y = y;
        size_hints.width = width;
        size_hints.height = height;
        size_hints.min_width = 300;
        size_hints.min_height = 300;

        XSetNormalHints (display, win, &size_hints);
        XStoreName(display, win, window_name);

        /* create graphics context */

        gc = XCreateGC (display, win, valuemask, &values);

        

         /* allocate the set of colors we will want to use for the drawing. */
        if ( DEBUG  )
         printf("6. Setting Display Attributes\n ");

        XSetBackground (display, gc, WhitePixel (display, screen));
        XSetForeground (display, gc, BlackPixel (display, screen));
        XSetLineAttributes (display, gc, 1, LineSolid, CapRound, JoinRound);

        

        attr[0].backing_store = Always;
        attr[0].backing_planes = 1;
        attr[0].backing_pixel = BlackPixel(display, screen);

        XChangeWindowAttributes(display, win, CWBackingStore | CWBackingPlanes | CWBackingPixel, attr);

        XMapWindow (display, win);
        XSync(display, 0);

        

        int white, black;
        int generic,process;
        int minc, maxc;

        if ( DEBUG  )
         printf("7. Computing the Numerical Value of Black and White Pixel\n");

        white = WhitePixel (display, screen);       /* color value for white */
        black = BlackPixel (display, screen);       /* color value for black */
        
        minc = (white > black) ? black : white;
        maxc = (white > black) ? white : black;
        
	/* Loop around each point and determine the color*/
	
	if ( DEBUG  )
         printf("8. Entering the Iterate function to compute the Color at each point\n");
	
	dim3 blocks(80,80,1);
	dim3 threads(10,10,1);
	iterate<<<blocks,threads>>>(maxc, minc, X_RESN, Y_RESN,TheK_device, max_iteration);
	hipDeviceSynchronize();

	if ( DEBUG  )
         printf("9. End of Iterate() Function.. exiting\n");

    hipError_t error;
    error= hipMemcpy(TheK, TheK_device, size, hipMemcpyDeviceToHost);
    if (DEBUG)
    {
    	printf("\nError in Copying: %s\n", hipGetErrorString(error));
    }
        
	if (DEBUG )
	{
	  printf("10. Value for white = %d\n",white);
	  printf("11. Value for black = %d\n",black);
	}  
	
    
    	clock_t t1, t2;
        t1=clock();
           
	if ( DEBUG  )
         printf("12. Loop Around each Pixel, Set the Color and Plot the Point\n ");
    
       for(i = 0; i < Y_RESN ; i++)
        {
                for(j=0; j < X_RESN; j++)
                 {
                        XSetForeground(display, gc, TheK[i*Y_RESN+j]  );
                        XDrawPoint (display, win, gc, j, i);
                 }

        }
	t2=clock();   
       
        if ( DEBUG  )
         printf("13. End of Code\n ");
       
    if (DEBUG)
      {
      	printf("Total Time Taken for Plotting= %f Seconds\n",((double)( t2 - t1 )/ (double) CLOCKS_PER_SEC ) );
      }
        sleep(5);
        XFlush (display);
        
	if ( DEBUG  )
         printf("14. Free the Assigned Memory\n");
	
	free(TheK);
	hipFree(TheK_device);

}